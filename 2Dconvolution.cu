#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>

// includes, project
#include "2Dconvolution.h"

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(int width, int height);
Matrix AllocateMatrix(int width, int height);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);
void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P);

////////////////////////////////////////////////////////////////////////////////
// Înmulțirea fără memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernel(Matrix M, Matrix N, Matrix P)
{

    float Cvalue = 0;
    //se calculeaza pozitia elementulu din matricea P
  	int row = blockIdx.y * blockDim.y + threadIdx.y;
  	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int i,j;
	//se retine in Cvalue suma dupa formula din enunt
	for(i=0;i<5;++i){
		for(j=0;j<5;j++){
			if((row+i-2>=0) && (col+j-2>=0) && (row+i-2<N.height) && (col+j-2<N.width) && (row<N.height) && (col < N.width))
				Cvalue += M.elements[i*5+j]*N.elements[(row+i-2)*N.width+col+j-2];
		}
	}
	//se adauga in P pe pozitia row*N.width+col valoarea sumei
  	P.elements[row * N.width + col] =(float)Cvalue;
}


////////////////////////////////////////////////////////////////////////////////
// Înmulțirea cu memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernelShared(Matrix M, Matrix N, Matrix P)
{

    //TODO: calculul rezultatului convoluției

}

////////////////////////////////////////////////////////////////////////////////
// Returnează 1 dacă matricele sunt ~ egale
////////////////////////////////////////////////////////////////////////////////
int CompareMatrices(Matrix A, Matrix B)
{
    int i;
    if(A.width != B.width || A.height != B.height || A.pitch != B.pitch)
        return 0;
    int size = A.width * A.height;
    for(i = 0; i < size; i++)
        if(fabs(A.elements[i] - B.elements[i]) > MAX_ERR)
            return 0;
    return 1;
}
void GenerateRandomMatrix(Matrix m)
{
    int i;
    int size = m.width * m.height;

    srand(time(NULL));

    for(i = 0; i < size; i++)
        m.elements[i] = rand() / (float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
    int width = 0, height = 0;
    FILE *f, *out, *grafic;
    if(argc < 2)
    {
        printf("Argumente prea puține, trimiteți id-ul testului care trebuie rulat\n");
        return 0;
    }
    char name[100];
    sprintf(name, "./tests/test_%s.txt", argv[1]);
    f = fopen(name, "r");
    out = fopen("out.txt", "a");
    grafic = fopen("grafic.txt", "a");
    fscanf(f, "%d%d", &width, &height);
    fprintf(grafic,"%d ",width*height);
    Matrix M;//kernel de pe host
    Matrix N;//matrice inițială de pe host
    Matrix P;//rezultat fără memorie partajată calculat pe GPU
    Matrix PS;//rezultatul cu memorie partajată calculat pe GPU
    
    M = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE);
    N = AllocateMatrix(width, height);        
    P = AllocateMatrix(width, height);
    PS = AllocateMatrix(width, height);
	
    GenerateRandomMatrix(M);
    GenerateRandomMatrix(N);
    
    // M * N pe device
    ConvolutionOnDevice(M, N, P);

    // M * N pe device cu memorie partajată
    ConvolutionOnDeviceShared(M, N, PS);
    // calculează rezultatul pe CPU pentru comparație
    Matrix reference = AllocateMatrix(P.width, P.height);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);
	fprintf(grafic,"%s","\n");
    // verifică dacă rezultatul obținut pe device este cel așteptat
    int res = CompareMatrices(reference, P);
    printf("Test global %s\n", (1 == res) ? "PASSED" : "FAILED");
    fprintf(out, "Test global %s %s\n", argv[1], (1 == res) ? "PASSED" : "FAILED");

    // verifică dacă rezultatul obținut pe device cu memorie partajată este cel așteptat
  //  int ress = CompareMatrices(reference, PS);
    int ress = CompareMatrices(reference, PS);
    printf("Test shared %s\n", (1 == ress) ? "PASSED" : "FAILED");
    fprintf(out, "Test shared %s %s\n", argv[1], (1 == ress) ? "PASSED" : "FAILED");
   
    // Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
    FreeMatrix(&PS);

    fclose(f);
    fclose(out);
    fclose(grafic);
    return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{

	/*am adaugat variabila FILE * grafic
	ce reprezinta fisierul de output pentru timp*/


    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device
	FILE *grafic;
	float timp;
	grafic = fopen("grafic.txt", "a");
    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    //alocare matricele de pe device
	Md = AllocateDeviceMatrix(M.width,M.height);
	Nd = AllocateDeviceMatrix(N.width,N.height);
	Pd = AllocateDeviceMatrix(N.width,N.height);

    //copiere date de pe host (M, N) pe device (MD, Nd)
    int size = M.width * M.height * sizeof(float);
	hipMemcpy( Md.elements, M.elements, size, hipMemcpyHostToDevice);
	size = N.width * N.height * sizeof(float);
	hipMemcpy( Nd.elements, N.elements, size, hipMemcpyHostToDevice);
   
    //setare configurație de rulare a kernelului
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
 	dim3 dimGrid((N.width + BLOCK_SIZE - 1) /dimBlock.x,(N.height + BLOCK_SIZE - 1) /dimBlock.y);
    sdkStartTimer(&kernelTime);
    
    //lansare în execuție a kernelului
    ConvolutionKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
    hipDeviceSynchronize(); 
    sdkStopTimer(&kernelTime);
    timp = sdkGetTimerValue(&kernelTime);
    printf ("Timp execuție kernel: %f ms\n", timp);
    fprintf(grafic,"%f ",timp);
    
    //copiere rezultat pe host
    size = N.width * N.height * sizeof(float);
    hipMemcpy( P.elements, Pd.elements, size, hipMemcpyDeviceToHost);
    
    //eliberarea memoriei matricelor de pe device
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
    fclose(grafic);
}

//nu am realizat varianta cu memory shared
void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P)
{
    //Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device
	FILE *grafic;
	float timp;
	grafic = fopen("grafic.txt", "a");
    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    //TODO: alocați matricele de pe device

    //TODO: copiați datele de pe host (M, N) pe device (MD, Nd)

    //TODO: setați configurația de rulare a kernelului

    sdkStartTimer(&kernelTime);
    //TODO: lansați în execuție kernelul    
    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    timp = sdkGetTimerValue(&kernelTime);
    printf ("Timp execuție kernel cu memorie partajată: %f ms\n", timp);
    fprintf(grafic,"%f ",timp);
    //TODO: copiaţi rezultatul pe host
    //TODO: eliberați memoria matricelor de pe device
    fclose(grafic);
}


// Alocă o matrice de dimensiune height*width pe device
Matrix AllocateDeviceMatrix(int width, int height)
{
	//TODO: alocați matricea și setați width, pitch și height
    Matrix m;
   	m.width = width;
   	m.height = height;
   	m.pitch = width;
   	size_t size = width * height * sizeof(float);
   	hipMalloc((void**)&m.elements,size);
    return m;
}

// Alocă matrice pe host de dimensiune height*width
Matrix AllocateMatrix(int width, int height)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;    
    M.elements = (float*) malloc(size*sizeof(float));
    return M;
}    

// Eliberează o matrice de pe device
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Eliberează o matrice de pe host
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}
